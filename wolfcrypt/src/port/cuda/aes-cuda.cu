#include "hip/hip_runtime.h"
/* aes.cu
 *
 * Copyright (C) 2006-2025 wolfSSL Inc.
 *
 * This file is part of wolfSSL.
 *
 * wolfSSL is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * wolfSSL is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1335, USA
 */

/*

DESCRIPTION
This library provides the interfaces to the Advanced Encryption Standard (AES)
for encrypting and decrypting data. AES is the standard known for a symmetric
block cipher mechanism that uses n-bit binary string parameter key with 128-bits,
192-bits, and 256-bits of key sizes.

*/
#ifdef HAVE_CONFIG_H
    #include <config.h>
#endif

#include <wolfssl/wolfcrypt/settings.h>
#include <wolfssl/wolfcrypt/error-crypt.h>

#include <wolfssl/wolfcrypt/aes.h>

#ifdef WOLFSSL_AESNI
#include <wmmintrin.h>
#include <emmintrin.h>
#include <smmintrin.h>
#endif /* WOLFSSL_AESNI */

#include <wolfssl/wolfcrypt/cpuid.h>

#ifdef WOLF_CRYPTO_CB
    #include <wolfssl/wolfcrypt/cryptocb.h>
#endif

#ifdef WOLFSSL_SECO_CAAM
#include <wolfssl/wolfcrypt/port/caam/wolfcaam.h>
#endif

#ifdef WOLFSSL_IMXRT_DCP
    #include <wolfssl/wolfcrypt/port/nxp/dcp_port.h>
#endif
#if defined(WOLFSSL_SE050) && defined(WOLFSSL_SE050_CRYPT)
    #include <wolfssl/wolfcrypt/port/nxp/se050_port.h>
#endif

#if defined(WOLFSSL_AES_SIV)
    #include <wolfssl/wolfcrypt/cmac.h>
#endif /* WOLFSSL_AES_SIV */

#if defined(WOLFSSL_HAVE_PSA) && !defined(WOLFSSL_PSA_NO_AES)
    #include <wolfssl/wolfcrypt/port/psa/psa.h>
#endif


#include <wolfssl/wolfcrypt/logging.h>

#ifdef NO_INLINE
    #include <wolfssl/wolfcrypt/misc.h>
#else
    #define WOLFSSL_MISC_INCLUDED
    #define WOLFSSL_HAVE_MIN
    #define WOLFSSL_HAVE_MAX
/*    #include <wolfcrypt/src/misc.c> */
#endif
/* This routine performs a left circular arithmetic shift of <x> by <y> value. */

extern "C" {

#if !defined(WOLFSSL_TI_CRYPT)

#define rotlFixed(x, y) ( (x << y) | (x >> (sizeof(x) * 8 - y)) )

/* This routine performs a right circular arithmetic shift of <x> by <y> value. */
#define rotrFixed(x, y) ( (x >> y) | (x << (sizeof(x) * 8 - y)) )

#ifdef WC_RC2

/* This routine performs a left circular arithmetic shift of <x> by <y> value */
static WC_INLINE word16 rotlFixed16(word16 x, word16 y)
{
    return (x << y) | (x >> (sizeof(x) * 8 - y));
}


/* This routine performs a right circular arithmetic shift of <x> by <y> value */
static WC_INLINE word16 rotrFixed16(word16 x, word16 y)
{
    return (x >> y) | (x << (sizeof(x) * 8 - y));
}

#endif /* WC_RC2 */

/* This routine performs a byte swap of 32-bit word value. */
#if defined(__CCRX__) && !defined(NO_INLINE) /* shortest version for CC-RX */
    #define ByteReverseWord32(value, outRef) ( *outRef = _builtin_revl(value) )
#else
__device__
static WC_INLINE word32 ByteReverseWord32(word32 value)
{
#ifdef PPC_INTRINSICS
    /* PPC: load reverse indexed instruction */
    return (word32)__lwbrx(&value,0);
#elif defined(__ICCARM__)
    return (word32)__REV(value);
#elif defined(KEIL_INTRINSICS)
    return (word32)__rev(value);
#elif defined(__CCRX__)
    return (word32)_builtin_revl(value);
#elif defined(WOLF_ALLOW_BUILTIN) && \
        defined(__GNUC_PREREQ) && __GNUC_PREREQ(4, 3)
    return (word32)__builtin_bswap32(value);
#elif defined(WOLFSSL_BYTESWAP32_ASM) && defined(__GNUC__) && \
      defined(__aarch64__)
    __asm__ volatile (
        "REV32 %0, %0  \n"
        : "+r" (value)
        :
    );
    return value;
#elif defined(WOLFSSL_BYTESWAP32_ASM) && defined(__GNUC__) && \
      (defined(__thumb__) || defined(__arm__))
    __asm__ volatile (
        "REV %0, %0  \n"
        : "+r" (value)
        :
    );
    return value;
#elif defined(FAST_ROTATE)
    /* 5 instructions with rotate instruction, 9 without */
    return (rotrFixed(value, 8U) & 0xff00ff00) |
           (rotlFixed(value, 8U) & 0x00ff00ff);
#else
    /* 6 instructions with rotate instruction, 8 without */
    value = ((value & 0xFF00FF00) >> 8) | ((value & 0x00FF00FF) << 8);
    return rotlFixed(value, 16U);
#endif
}
#endif /* ! (__CCRX__ && !NO_INLINE) */

#if defined(STM32_CRYPTO)
#elif defined(HAVE_COLDFIRE_SEC)
#elif defined(FREESCALE_LTC)
#elif defined(FREESCALE_MMCAU)
#elif defined(WOLFSSL_PIC32MZ_CRYPT)
#elif defined(WOLFSSL_NRF51_AES)
#elif defined(WOLFSSL_ESP32_CRYPT) && \
     !defined(NO_WOLFSSL_ESP32_CRYPT_AES)
#elif defined(WOLFSSL_AESNI)
#elif (defined(WOLFSSL_IMX6_CAAM) && !defined(NO_IMX6_CAAM_AES) \
        && !defined(WOLFSSL_QNX_CAAM)) || \
      ((defined(WOLFSSL_AFALG) || defined(WOLFSSL_DEVCRYPTO_AES)) && \
        defined(HAVE_AESCCM))
#elif defined(WOLFSSL_AFALG)
    /* implemented in wolfcrypt/src/port/af_alg/afalg_aes.c */

#elif defined(WOLFSSL_DEVCRYPTO_AES)
    /* implemented in wolfcrypt/src/port/devcrypto/devcrypto_aes.c */

#elif defined(WOLFSSL_SCE) && !defined(WOLFSSL_SCE_NO_AES)
#elif defined(WOLFSSL_KCAPI_AES)
#elif defined(WOLFSSL_HAVE_PSA) && !defined(WOLFSSL_PSA_NO_AES)
/* implemented in wolfcrypt/src/port/psa/psa_aes.c */

#else

    /* using wolfCrypt software implementation */
    #define NEED_AES_TABLES
#endif

#if !defined(NO_AES) && !defined(WOLFSSL_TI_CRYPT) && !defined(WOLFSSL_ARMASM) && \
    defined(NEED_AES_TABLES) && (defined(HAVE_AES_CBC) || defined(WOLFSSL_AES_DIRECT) || defined(HAVE_AESCCM) || defined(HAVE_AESGCM)) && \
    defined(HAVE_CUDA)

#define GETBYTE(x, y) (word32)((byte)((x) >> (8 * (y))))

#ifndef WC_CACHE_LINE_SZ
    #if defined(__x86_64__) || defined(_M_X64) || \
       (defined(__ILP32__) && (__ILP32__ >= 1))
        #define WC_CACHE_LINE_SZ 64
    #else
        /* default cache line size */
        #define WC_CACHE_LINE_SZ 32
    #endif
#endif

#if WC_CACHE_LINE_SZ == 128
    #define WC_CACHE_LINE_BITS      5
    #define WC_CACHE_LINE_MASK_HI   0xe0
    #define WC_CACHE_LINE_MASK_LO   0x1f
    #define WC_CACHE_LINE_ADD       0x20
#elif WC_CACHE_LINE_SZ == 64
    #define WC_CACHE_LINE_BITS      4
    #define WC_CACHE_LINE_MASK_HI   0xf0
    #define WC_CACHE_LINE_MASK_LO   0x0f
    #define WC_CACHE_LINE_ADD       0x10
#elif WC_CACHE_LINE_SZ == 32
    #define WC_CACHE_LINE_BITS      3
    #define WC_CACHE_LINE_MASK_HI   0xf8
    #define WC_CACHE_LINE_MASK_LO   0x07
    #define WC_CACHE_LINE_ADD       0x08
#elif WC_CACHE_LINE_SZ == 16
    #define WC_CACHE_LINE_BITS      2
    #define WC_CACHE_LINE_MASK_HI   0xfc
    #define WC_CACHE_LINE_MASK_LO   0x03
    #define WC_CACHE_LINE_ADD       0x04
#else
    #error Cache line size not supported
#endif

#ifndef WOLFSSL_AES_SMALL_TABLES
__device__
const FLASH_QUALIFIER word32 Te_CUDA[4][256] = {
{
    0xc66363a5U, 0xf87c7c84U, 0xee777799U, 0xf67b7b8dU,
    0xfff2f20dU, 0xd66b6bbdU, 0xde6f6fb1U, 0x91c5c554U,
    0x60303050U, 0x02010103U, 0xce6767a9U, 0x562b2b7dU,
    0xe7fefe19U, 0xb5d7d762U, 0x4dababe6U, 0xec76769aU,
    0x8fcaca45U, 0x1f82829dU, 0x89c9c940U, 0xfa7d7d87U,
    0xeffafa15U, 0xb25959ebU, 0x8e4747c9U, 0xfbf0f00bU,
    0x41adadecU, 0xb3d4d467U, 0x5fa2a2fdU, 0x45afafeaU,
    0x239c9cbfU, 0x53a4a4f7U, 0xe4727296U, 0x9bc0c05bU,
    0x75b7b7c2U, 0xe1fdfd1cU, 0x3d9393aeU, 0x4c26266aU,
    0x6c36365aU, 0x7e3f3f41U, 0xf5f7f702U, 0x83cccc4fU,
    0x6834345cU, 0x51a5a5f4U, 0xd1e5e534U, 0xf9f1f108U,
    0xe2717193U, 0xabd8d873U, 0x62313153U, 0x2a15153fU,
    0x0804040cU, 0x95c7c752U, 0x46232365U, 0x9dc3c35eU,
    0x30181828U, 0x379696a1U, 0x0a05050fU, 0x2f9a9ab5U,
    0x0e070709U, 0x24121236U, 0x1b80809bU, 0xdfe2e23dU,
    0xcdebeb26U, 0x4e272769U, 0x7fb2b2cdU, 0xea75759fU,
    0x1209091bU, 0x1d83839eU, 0x582c2c74U, 0x341a1a2eU,
    0x361b1b2dU, 0xdc6e6eb2U, 0xb45a5aeeU, 0x5ba0a0fbU,
    0xa45252f6U, 0x763b3b4dU, 0xb7d6d661U, 0x7db3b3ceU,
    0x5229297bU, 0xdde3e33eU, 0x5e2f2f71U, 0x13848497U,
    0xa65353f5U, 0xb9d1d168U, 0x00000000U, 0xc1eded2cU,
    0x40202060U, 0xe3fcfc1fU, 0x79b1b1c8U, 0xb65b5bedU,
    0xd46a6abeU, 0x8dcbcb46U, 0x67bebed9U, 0x7239394bU,
    0x944a4adeU, 0x984c4cd4U, 0xb05858e8U, 0x85cfcf4aU,
    0xbbd0d06bU, 0xc5efef2aU, 0x4faaaae5U, 0xedfbfb16U,
    0x864343c5U, 0x9a4d4dd7U, 0x66333355U, 0x11858594U,
    0x8a4545cfU, 0xe9f9f910U, 0x04020206U, 0xfe7f7f81U,
    0xa05050f0U, 0x783c3c44U, 0x259f9fbaU, 0x4ba8a8e3U,
    0xa25151f3U, 0x5da3a3feU, 0x804040c0U, 0x058f8f8aU,
    0x3f9292adU, 0x219d9dbcU, 0x70383848U, 0xf1f5f504U,
    0x63bcbcdfU, 0x77b6b6c1U, 0xafdada75U, 0x42212163U,
    0x20101030U, 0xe5ffff1aU, 0xfdf3f30eU, 0xbfd2d26dU,
    0x81cdcd4cU, 0x180c0c14U, 0x26131335U, 0xc3ecec2fU,
    0xbe5f5fe1U, 0x359797a2U, 0x884444ccU, 0x2e171739U,
    0x93c4c457U, 0x55a7a7f2U, 0xfc7e7e82U, 0x7a3d3d47U,
    0xc86464acU, 0xba5d5de7U, 0x3219192bU, 0xe6737395U,
    0xc06060a0U, 0x19818198U, 0x9e4f4fd1U, 0xa3dcdc7fU,
    0x44222266U, 0x542a2a7eU, 0x3b9090abU, 0x0b888883U,
    0x8c4646caU, 0xc7eeee29U, 0x6bb8b8d3U, 0x2814143cU,
    0xa7dede79U, 0xbc5e5ee2U, 0x160b0b1dU, 0xaddbdb76U,
    0xdbe0e03bU, 0x64323256U, 0x743a3a4eU, 0x140a0a1eU,
    0x924949dbU, 0x0c06060aU, 0x4824246cU, 0xb85c5ce4U,
    0x9fc2c25dU, 0xbdd3d36eU, 0x43acacefU, 0xc46262a6U,
    0x399191a8U, 0x319595a4U, 0xd3e4e437U, 0xf279798bU,
    0xd5e7e732U, 0x8bc8c843U, 0x6e373759U, 0xda6d6db7U,
    0x018d8d8cU, 0xb1d5d564U, 0x9c4e4ed2U, 0x49a9a9e0U,
    0xd86c6cb4U, 0xac5656faU, 0xf3f4f407U, 0xcfeaea25U,
    0xca6565afU, 0xf47a7a8eU, 0x47aeaee9U, 0x10080818U,
    0x6fbabad5U, 0xf0787888U, 0x4a25256fU, 0x5c2e2e72U,
    0x381c1c24U, 0x57a6a6f1U, 0x73b4b4c7U, 0x97c6c651U,
    0xcbe8e823U, 0xa1dddd7cU, 0xe874749cU, 0x3e1f1f21U,
    0x964b4bddU, 0x61bdbddcU, 0x0d8b8b86U, 0x0f8a8a85U,
    0xe0707090U, 0x7c3e3e42U, 0x71b5b5c4U, 0xcc6666aaU,
    0x904848d8U, 0x06030305U, 0xf7f6f601U, 0x1c0e0e12U,
    0xc26161a3U, 0x6a35355fU, 0xae5757f9U, 0x69b9b9d0U,
    0x17868691U, 0x99c1c158U, 0x3a1d1d27U, 0x279e9eb9U,
    0xd9e1e138U, 0xebf8f813U, 0x2b9898b3U, 0x22111133U,
    0xd26969bbU, 0xa9d9d970U, 0x078e8e89U, 0x339494a7U,
    0x2d9b9bb6U, 0x3c1e1e22U, 0x15878792U, 0xc9e9e920U,
    0x87cece49U, 0xaa5555ffU, 0x50282878U, 0xa5dfdf7aU,
    0x038c8c8fU, 0x59a1a1f8U, 0x09898980U, 0x1a0d0d17U,
    0x65bfbfdaU, 0xd7e6e631U, 0x844242c6U, 0xd06868b8U,
    0x824141c3U, 0x299999b0U, 0x5a2d2d77U, 0x1e0f0f11U,
    0x7bb0b0cbU, 0xa85454fcU, 0x6dbbbbd6U, 0x2c16163aU,
},
{
    0xa5c66363U, 0x84f87c7cU, 0x99ee7777U, 0x8df67b7bU,
    0x0dfff2f2U, 0xbdd66b6bU, 0xb1de6f6fU, 0x5491c5c5U,
    0x50603030U, 0x03020101U, 0xa9ce6767U, 0x7d562b2bU,
    0x19e7fefeU, 0x62b5d7d7U, 0xe64dababU, 0x9aec7676U,
    0x458fcacaU, 0x9d1f8282U, 0x4089c9c9U, 0x87fa7d7dU,
    0x15effafaU, 0xebb25959U, 0xc98e4747U, 0x0bfbf0f0U,
    0xec41adadU, 0x67b3d4d4U, 0xfd5fa2a2U, 0xea45afafU,
    0xbf239c9cU, 0xf753a4a4U, 0x96e47272U, 0x5b9bc0c0U,
    0xc275b7b7U, 0x1ce1fdfdU, 0xae3d9393U, 0x6a4c2626U,
    0x5a6c3636U, 0x417e3f3fU, 0x02f5f7f7U, 0x4f83ccccU,
    0x5c683434U, 0xf451a5a5U, 0x34d1e5e5U, 0x08f9f1f1U,
    0x93e27171U, 0x73abd8d8U, 0x53623131U, 0x3f2a1515U,
    0x0c080404U, 0x5295c7c7U, 0x65462323U, 0x5e9dc3c3U,
    0x28301818U, 0xa1379696U, 0x0f0a0505U, 0xb52f9a9aU,
    0x090e0707U, 0x36241212U, 0x9b1b8080U, 0x3ddfe2e2U,
    0x26cdebebU, 0x694e2727U, 0xcd7fb2b2U, 0x9fea7575U,
    0x1b120909U, 0x9e1d8383U, 0x74582c2cU, 0x2e341a1aU,
    0x2d361b1bU, 0xb2dc6e6eU, 0xeeb45a5aU, 0xfb5ba0a0U,
    0xf6a45252U, 0x4d763b3bU, 0x61b7d6d6U, 0xce7db3b3U,
    0x7b522929U, 0x3edde3e3U, 0x715e2f2fU, 0x97138484U,
    0xf5a65353U, 0x68b9d1d1U, 0x00000000U, 0x2cc1ededU,
    0x60402020U, 0x1fe3fcfcU, 0xc879b1b1U, 0xedb65b5bU,
    0xbed46a6aU, 0x468dcbcbU, 0xd967bebeU, 0x4b723939U,
    0xde944a4aU, 0xd4984c4cU, 0xe8b05858U, 0x4a85cfcfU,
    0x6bbbd0d0U, 0x2ac5efefU, 0xe54faaaaU, 0x16edfbfbU,
    0xc5864343U, 0xd79a4d4dU, 0x55663333U, 0x94118585U,
    0xcf8a4545U, 0x10e9f9f9U, 0x06040202U, 0x81fe7f7fU,
    0xf0a05050U, 0x44783c3cU, 0xba259f9fU, 0xe34ba8a8U,
    0xf3a25151U, 0xfe5da3a3U, 0xc0804040U, 0x8a058f8fU,
    0xad3f9292U, 0xbc219d9dU, 0x48703838U, 0x04f1f5f5U,
    0xdf63bcbcU, 0xc177b6b6U, 0x75afdadaU, 0x63422121U,
    0x30201010U, 0x1ae5ffffU, 0x0efdf3f3U, 0x6dbfd2d2U,
    0x4c81cdcdU, 0x14180c0cU, 0x35261313U, 0x2fc3ececU,
    0xe1be5f5fU, 0xa2359797U, 0xcc884444U, 0x392e1717U,
    0x5793c4c4U, 0xf255a7a7U, 0x82fc7e7eU, 0x477a3d3dU,
    0xacc86464U, 0xe7ba5d5dU, 0x2b321919U, 0x95e67373U,
    0xa0c06060U, 0x98198181U, 0xd19e4f4fU, 0x7fa3dcdcU,
    0x66442222U, 0x7e542a2aU, 0xab3b9090U, 0x830b8888U,
    0xca8c4646U, 0x29c7eeeeU, 0xd36bb8b8U, 0x3c281414U,
    0x79a7dedeU, 0xe2bc5e5eU, 0x1d160b0bU, 0x76addbdbU,
    0x3bdbe0e0U, 0x56643232U, 0x4e743a3aU, 0x1e140a0aU,
    0xdb924949U, 0x0a0c0606U, 0x6c482424U, 0xe4b85c5cU,
    0x5d9fc2c2U, 0x6ebdd3d3U, 0xef43acacU, 0xa6c46262U,
    0xa8399191U, 0xa4319595U, 0x37d3e4e4U, 0x8bf27979U,
    0x32d5e7e7U, 0x438bc8c8U, 0x596e3737U, 0xb7da6d6dU,
    0x8c018d8dU, 0x64b1d5d5U, 0xd29c4e4eU, 0xe049a9a9U,
    0xb4d86c6cU, 0xfaac5656U, 0x07f3f4f4U, 0x25cfeaeaU,
    0xafca6565U, 0x8ef47a7aU, 0xe947aeaeU, 0x18100808U,
    0xd56fbabaU, 0x88f07878U, 0x6f4a2525U, 0x725c2e2eU,
    0x24381c1cU, 0xf157a6a6U, 0xc773b4b4U, 0x5197c6c6U,
    0x23cbe8e8U, 0x7ca1ddddU, 0x9ce87474U, 0x213e1f1fU,
    0xdd964b4bU, 0xdc61bdbdU, 0x860d8b8bU, 0x850f8a8aU,
    0x90e07070U, 0x427c3e3eU, 0xc471b5b5U, 0xaacc6666U,
    0xd8904848U, 0x05060303U, 0x01f7f6f6U, 0x121c0e0eU,
    0xa3c26161U, 0x5f6a3535U, 0xf9ae5757U, 0xd069b9b9U,
    0x91178686U, 0x5899c1c1U, 0x273a1d1dU, 0xb9279e9eU,
    0x38d9e1e1U, 0x13ebf8f8U, 0xb32b9898U, 0x33221111U,
    0xbbd26969U, 0x70a9d9d9U, 0x89078e8eU, 0xa7339494U,
    0xb62d9b9bU, 0x223c1e1eU, 0x92158787U, 0x20c9e9e9U,
    0x4987ceceU, 0xffaa5555U, 0x78502828U, 0x7aa5dfdfU,
    0x8f038c8cU, 0xf859a1a1U, 0x80098989U, 0x171a0d0dU,
    0xda65bfbfU, 0x31d7e6e6U, 0xc6844242U, 0xb8d06868U,
    0xc3824141U, 0xb0299999U, 0x775a2d2dU, 0x111e0f0fU,
    0xcb7bb0b0U, 0xfca85454U, 0xd66dbbbbU, 0x3a2c1616U,
},
{
    0x63a5c663U, 0x7c84f87cU, 0x7799ee77U, 0x7b8df67bU,
    0xf20dfff2U, 0x6bbdd66bU, 0x6fb1de6fU, 0xc55491c5U,
    0x30506030U, 0x01030201U, 0x67a9ce67U, 0x2b7d562bU,
    0xfe19e7feU, 0xd762b5d7U, 0xabe64dabU, 0x769aec76U,
    0xca458fcaU, 0x829d1f82U, 0xc94089c9U, 0x7d87fa7dU,
    0xfa15effaU, 0x59ebb259U, 0x47c98e47U, 0xf00bfbf0U,
    0xadec41adU, 0xd467b3d4U, 0xa2fd5fa2U, 0xafea45afU,
    0x9cbf239cU, 0xa4f753a4U, 0x7296e472U, 0xc05b9bc0U,
    0xb7c275b7U, 0xfd1ce1fdU, 0x93ae3d93U, 0x266a4c26U,
    0x365a6c36U, 0x3f417e3fU, 0xf702f5f7U, 0xcc4f83ccU,
    0x345c6834U, 0xa5f451a5U, 0xe534d1e5U, 0xf108f9f1U,
    0x7193e271U, 0xd873abd8U, 0x31536231U, 0x153f2a15U,
    0x040c0804U, 0xc75295c7U, 0x23654623U, 0xc35e9dc3U,
    0x18283018U, 0x96a13796U, 0x050f0a05U, 0x9ab52f9aU,
    0x07090e07U, 0x12362412U, 0x809b1b80U, 0xe23ddfe2U,
    0xeb26cdebU, 0x27694e27U, 0xb2cd7fb2U, 0x759fea75U,
    0x091b1209U, 0x839e1d83U, 0x2c74582cU, 0x1a2e341aU,
    0x1b2d361bU, 0x6eb2dc6eU, 0x5aeeb45aU, 0xa0fb5ba0U,
    0x52f6a452U, 0x3b4d763bU, 0xd661b7d6U, 0xb3ce7db3U,
    0x297b5229U, 0xe33edde3U, 0x2f715e2fU, 0x84971384U,
    0x53f5a653U, 0xd168b9d1U, 0x00000000U, 0xed2cc1edU,
    0x20604020U, 0xfc1fe3fcU, 0xb1c879b1U, 0x5bedb65bU,
    0x6abed46aU, 0xcb468dcbU, 0xbed967beU, 0x394b7239U,
    0x4ade944aU, 0x4cd4984cU, 0x58e8b058U, 0xcf4a85cfU,
    0xd06bbbd0U, 0xef2ac5efU, 0xaae54faaU, 0xfb16edfbU,
    0x43c58643U, 0x4dd79a4dU, 0x33556633U, 0x85941185U,
    0x45cf8a45U, 0xf910e9f9U, 0x02060402U, 0x7f81fe7fU,
    0x50f0a050U, 0x3c44783cU, 0x9fba259fU, 0xa8e34ba8U,
    0x51f3a251U, 0xa3fe5da3U, 0x40c08040U, 0x8f8a058fU,
    0x92ad3f92U, 0x9dbc219dU, 0x38487038U, 0xf504f1f5U,
    0xbcdf63bcU, 0xb6c177b6U, 0xda75afdaU, 0x21634221U,
    0x10302010U, 0xff1ae5ffU, 0xf30efdf3U, 0xd26dbfd2U,
    0xcd4c81cdU, 0x0c14180cU, 0x13352613U, 0xec2fc3ecU,
    0x5fe1be5fU, 0x97a23597U, 0x44cc8844U, 0x17392e17U,
    0xc45793c4U, 0xa7f255a7U, 0x7e82fc7eU, 0x3d477a3dU,
    0x64acc864U, 0x5de7ba5dU, 0x192b3219U, 0x7395e673U,
    0x60a0c060U, 0x81981981U, 0x4fd19e4fU, 0xdc7fa3dcU,
    0x22664422U, 0x2a7e542aU, 0x90ab3b90U, 0x88830b88U,
    0x46ca8c46U, 0xee29c7eeU, 0xb8d36bb8U, 0x143c2814U,
    0xde79a7deU, 0x5ee2bc5eU, 0x0b1d160bU, 0xdb76addbU,
    0xe03bdbe0U, 0x32566432U, 0x3a4e743aU, 0x0a1e140aU,
    0x49db9249U, 0x060a0c06U, 0x246c4824U, 0x5ce4b85cU,
    0xc25d9fc2U, 0xd36ebdd3U, 0xacef43acU, 0x62a6c462U,
    0x91a83991U, 0x95a43195U, 0xe437d3e4U, 0x798bf279U,
    0xe732d5e7U, 0xc8438bc8U, 0x37596e37U, 0x6db7da6dU,
    0x8d8c018dU, 0xd564b1d5U, 0x4ed29c4eU, 0xa9e049a9U,
    0x6cb4d86cU, 0x56faac56U, 0xf407f3f4U, 0xea25cfeaU,
    0x65afca65U, 0x7a8ef47aU, 0xaee947aeU, 0x08181008U,
    0xbad56fbaU, 0x7888f078U, 0x256f4a25U, 0x2e725c2eU,
    0x1c24381cU, 0xa6f157a6U, 0xb4c773b4U, 0xc65197c6U,
    0xe823cbe8U, 0xdd7ca1ddU, 0x749ce874U, 0x1f213e1fU,
    0x4bdd964bU, 0xbddc61bdU, 0x8b860d8bU, 0x8a850f8aU,
    0x7090e070U, 0x3e427c3eU, 0xb5c471b5U, 0x66aacc66U,
    0x48d89048U, 0x03050603U, 0xf601f7f6U, 0x0e121c0eU,
    0x61a3c261U, 0x355f6a35U, 0x57f9ae57U, 0xb9d069b9U,
    0x86911786U, 0xc15899c1U, 0x1d273a1dU, 0x9eb9279eU,
    0xe138d9e1U, 0xf813ebf8U, 0x98b32b98U, 0x11332211U,
    0x69bbd269U, 0xd970a9d9U, 0x8e89078eU, 0x94a73394U,
    0x9bb62d9bU, 0x1e223c1eU, 0x87921587U, 0xe920c9e9U,
    0xce4987ceU, 0x55ffaa55U, 0x28785028U, 0xdf7aa5dfU,
    0x8c8f038cU, 0xa1f859a1U, 0x89800989U, 0x0d171a0dU,
    0xbfda65bfU, 0xe631d7e6U, 0x42c68442U, 0x68b8d068U,
    0x41c38241U, 0x99b02999U, 0x2d775a2dU, 0x0f111e0fU,
    0xb0cb7bb0U, 0x54fca854U, 0xbbd66dbbU, 0x163a2c16U,
},
{
    0x6363a5c6U, 0x7c7c84f8U, 0x777799eeU, 0x7b7b8df6U,
    0xf2f20dffU, 0x6b6bbdd6U, 0x6f6fb1deU, 0xc5c55491U,
    0x30305060U, 0x01010302U, 0x6767a9ceU, 0x2b2b7d56U,
    0xfefe19e7U, 0xd7d762b5U, 0xababe64dU, 0x76769aecU,
    0xcaca458fU, 0x82829d1fU, 0xc9c94089U, 0x7d7d87faU,
    0xfafa15efU, 0x5959ebb2U, 0x4747c98eU, 0xf0f00bfbU,
    0xadadec41U, 0xd4d467b3U, 0xa2a2fd5fU, 0xafafea45U,
    0x9c9cbf23U, 0xa4a4f753U, 0x727296e4U, 0xc0c05b9bU,
    0xb7b7c275U, 0xfdfd1ce1U, 0x9393ae3dU, 0x26266a4cU,
    0x36365a6cU, 0x3f3f417eU, 0xf7f702f5U, 0xcccc4f83U,
    0x34345c68U, 0xa5a5f451U, 0xe5e534d1U, 0xf1f108f9U,
    0x717193e2U, 0xd8d873abU, 0x31315362U, 0x15153f2aU,
    0x04040c08U, 0xc7c75295U, 0x23236546U, 0xc3c35e9dU,
    0x18182830U, 0x9696a137U, 0x05050f0aU, 0x9a9ab52fU,
    0x0707090eU, 0x12123624U, 0x80809b1bU, 0xe2e23ddfU,
    0xebeb26cdU, 0x2727694eU, 0xb2b2cd7fU, 0x75759feaU,
    0x09091b12U, 0x83839e1dU, 0x2c2c7458U, 0x1a1a2e34U,
    0x1b1b2d36U, 0x6e6eb2dcU, 0x5a5aeeb4U, 0xa0a0fb5bU,
    0x5252f6a4U, 0x3b3b4d76U, 0xd6d661b7U, 0xb3b3ce7dU,
    0x29297b52U, 0xe3e33eddU, 0x2f2f715eU, 0x84849713U,
    0x5353f5a6U, 0xd1d168b9U, 0x00000000U, 0xeded2cc1U,
    0x20206040U, 0xfcfc1fe3U, 0xb1b1c879U, 0x5b5bedb6U,
    0x6a6abed4U, 0xcbcb468dU, 0xbebed967U, 0x39394b72U,
    0x4a4ade94U, 0x4c4cd498U, 0x5858e8b0U, 0xcfcf4a85U,
    0xd0d06bbbU, 0xefef2ac5U, 0xaaaae54fU, 0xfbfb16edU,
    0x4343c586U, 0x4d4dd79aU, 0x33335566U, 0x85859411U,
    0x4545cf8aU, 0xf9f910e9U, 0x02020604U, 0x7f7f81feU,
    0x5050f0a0U, 0x3c3c4478U, 0x9f9fba25U, 0xa8a8e34bU,
    0x5151f3a2U, 0xa3a3fe5dU, 0x4040c080U, 0x8f8f8a05U,
    0x9292ad3fU, 0x9d9dbc21U, 0x38384870U, 0xf5f504f1U,
    0xbcbcdf63U, 0xb6b6c177U, 0xdada75afU, 0x21216342U,
    0x10103020U, 0xffff1ae5U, 0xf3f30efdU, 0xd2d26dbfU,
    0xcdcd4c81U, 0x0c0c1418U, 0x13133526U, 0xecec2fc3U,
    0x5f5fe1beU, 0x9797a235U, 0x4444cc88U, 0x1717392eU,
    0xc4c45793U, 0xa7a7f255U, 0x7e7e82fcU, 0x3d3d477aU,
    0x6464acc8U, 0x5d5de7baU, 0x19192b32U, 0x737395e6U,
    0x6060a0c0U, 0x81819819U, 0x4f4fd19eU, 0xdcdc7fa3U,
    0x22226644U, 0x2a2a7e54U, 0x9090ab3bU, 0x8888830bU,
    0x4646ca8cU, 0xeeee29c7U, 0xb8b8d36bU, 0x14143c28U,
    0xdede79a7U, 0x5e5ee2bcU, 0x0b0b1d16U, 0xdbdb76adU,
    0xe0e03bdbU, 0x32325664U, 0x3a3a4e74U, 0x0a0a1e14U,
    0x4949db92U, 0x06060a0cU, 0x24246c48U, 0x5c5ce4b8U,
    0xc2c25d9fU, 0xd3d36ebdU, 0xacacef43U, 0x6262a6c4U,
    0x9191a839U, 0x9595a431U, 0xe4e437d3U, 0x79798bf2U,
    0xe7e732d5U, 0xc8c8438bU, 0x3737596eU, 0x6d6db7daU,
    0x8d8d8c01U, 0xd5d564b1U, 0x4e4ed29cU, 0xa9a9e049U,
    0x6c6cb4d8U, 0x5656faacU, 0xf4f407f3U, 0xeaea25cfU,
    0x6565afcaU, 0x7a7a8ef4U, 0xaeaee947U, 0x08081810U,
    0xbabad56fU, 0x787888f0U, 0x25256f4aU, 0x2e2e725cU,
    0x1c1c2438U, 0xa6a6f157U, 0xb4b4c773U, 0xc6c65197U,
    0xe8e823cbU, 0xdddd7ca1U, 0x74749ce8U, 0x1f1f213eU,
    0x4b4bdd96U, 0xbdbddc61U, 0x8b8b860dU, 0x8a8a850fU,
    0x707090e0U, 0x3e3e427cU, 0xb5b5c471U, 0x6666aaccU,
    0x4848d890U, 0x03030506U, 0xf6f601f7U, 0x0e0e121cU,
    0x6161a3c2U, 0x35355f6aU, 0x5757f9aeU, 0xb9b9d069U,
    0x86869117U, 0xc1c15899U, 0x1d1d273aU, 0x9e9eb927U,
    0xe1e138d9U, 0xf8f813ebU, 0x9898b32bU, 0x11113322U,
    0x6969bbd2U, 0xd9d970a9U, 0x8e8e8907U, 0x9494a733U,
    0x9b9bb62dU, 0x1e1e223cU, 0x87879215U, 0xe9e920c9U,
    0xcece4987U, 0x5555ffaaU, 0x28287850U, 0xdfdf7aa5U,
    0x8c8c8f03U, 0xa1a1f859U, 0x89898009U, 0x0d0d171aU,
    0xbfbfda65U, 0xe6e631d7U, 0x4242c684U, 0x6868b8d0U,
    0x4141c382U, 0x9999b029U, 0x2d2d775aU, 0x0f0f111eU,
    0xb0b0cb7bU, 0x5454fca8U, 0xbbbbd66dU, 0x16163a2cU,
}
};


__device__
static word32 GetTable(const word32* t, byte o)
{
    word32 e = 0;
#if WC_CACHE_LINE_SZ == 64
  byte hi = o & 0xf0;
  byte lo = o & 0x0f;

  e  = t[lo + 0x00] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0x10] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0x20] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0x30] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0x40] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0x50] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0x60] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0x70] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0x80] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0x90] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0xa0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0xb0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0xc0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0xd0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0xe0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  e |= t[lo + 0xf0] & ((word32)0 - (((word32)hi - 0x01) >> 31));
#else
  int i;
  byte hi = o & WC_CACHE_LINE_MASK_HI;
  byte lo = o & WC_CACHE_LINE_MASK_LO;

  for (i = 0; i < 256; i += (1 << WC_CACHE_LINE_BITS)) {
      e |= t[lo + i] & ((word32)0 - (((word32)hi - 0x01) >> 31));
      hi -= WC_CACHE_LINE_ADD;
  }
#endif
  return e;
}

__device__
static void GetTable_Multi(const word32* t, word32* t0, byte o0,
  word32* t1, byte o1, word32* t2, byte o2, word32* t3, byte o3)
{
  word32 e0 = 0;
  word32 e1 = 0;
  word32 e2 = 0;
  word32 e3 = 0;
  byte hi0 = o0 & WC_CACHE_LINE_MASK_HI;
  byte lo0 = o0 & WC_CACHE_LINE_MASK_LO;
  byte hi1 = o1 & WC_CACHE_LINE_MASK_HI;
  byte lo1 = o1 & WC_CACHE_LINE_MASK_LO;
  byte hi2 = o2 & WC_CACHE_LINE_MASK_HI;
  byte lo2 = o2 & WC_CACHE_LINE_MASK_LO;
  byte hi3 = o3 & WC_CACHE_LINE_MASK_HI;
  byte lo3 = o3 & WC_CACHE_LINE_MASK_LO;
  int i;

  for (i = 0; i < 256; i += (1 << WC_CACHE_LINE_BITS)) {
      e0 |= t[lo0 + i] & ((word32)0 - (((word32)hi0 - 0x01) >> 31));
      hi0 -= WC_CACHE_LINE_ADD;
      e1 |= t[lo1 + i] & ((word32)0 - (((word32)hi1 - 0x01) >> 31));
      hi1 -= WC_CACHE_LINE_ADD;
      e2 |= t[lo2 + i] & ((word32)0 - (((word32)hi2 - 0x01) >> 31));
      hi2 -= WC_CACHE_LINE_ADD;
      e3 |= t[lo3 + i] & ((word32)0 - (((word32)hi3 - 0x01) >> 31));
      hi3 -= WC_CACHE_LINE_ADD;
  }
  *t0 = e0;
  *t1 = e1;
  *t2 = e2;
  *t3 = e3;
}

/* load 4 Te Tables into cache by cache line stride */
__device__
static WARN_UNUSED_RESULT WC_INLINE word32 PreFetchTe(void)
{
#ifndef WOLFSSL_AES_TOUCH_LINES
    word32 x = 0;
    int i,j;

    for (i = 0; i < 4; i++) {
        /* 256 elements, each one is 4 bytes */
        for (j = 0; j < 256; j += WC_CACHE_LINE_SZ/4) {
            x &= Te_CUDA[i][j];
        }
    }
    return x;
#else
    return 0;
#endif
}
#else
__device__ static const byte Tsbox[256] = {
    0x63U, 0x7cU, 0x77U, 0x7bU, 0xf2U, 0x6bU, 0x6fU, 0xc5U,
    0x30U, 0x01U, 0x67U, 0x2bU, 0xfeU, 0xd7U, 0xabU, 0x76U,
    0xcaU, 0x82U, 0xc9U, 0x7dU, 0xfaU, 0x59U, 0x47U, 0xf0U,
    0xadU, 0xd4U, 0xa2U, 0xafU, 0x9cU, 0xa4U, 0x72U, 0xc0U,
    0xb7U, 0xfdU, 0x93U, 0x26U, 0x36U, 0x3fU, 0xf7U, 0xccU,
    0x34U, 0xa5U, 0xe5U, 0xf1U, 0x71U, 0xd8U, 0x31U, 0x15U,
    0x04U, 0xc7U, 0x23U, 0xc3U, 0x18U, 0x96U, 0x05U, 0x9aU,
    0x07U, 0x12U, 0x80U, 0xe2U, 0xebU, 0x27U, 0xb2U, 0x75U,
    0x09U, 0x83U, 0x2cU, 0x1aU, 0x1bU, 0x6eU, 0x5aU, 0xa0U,
    0x52U, 0x3bU, 0xd6U, 0xb3U, 0x29U, 0xe3U, 0x2fU, 0x84U,
    0x53U, 0xd1U, 0x00U, 0xedU, 0x20U, 0xfcU, 0xb1U, 0x5bU,
    0x6aU, 0xcbU, 0xbeU, 0x39U, 0x4aU, 0x4cU, 0x58U, 0xcfU,
    0xd0U, 0xefU, 0xaaU, 0xfbU, 0x43U, 0x4dU, 0x33U, 0x85U,
    0x45U, 0xf9U, 0x02U, 0x7fU, 0x50U, 0x3cU, 0x9fU, 0xa8U,
    0x51U, 0xa3U, 0x40U, 0x8fU, 0x92U, 0x9dU, 0x38U, 0xf5U,
    0xbcU, 0xb6U, 0xdaU, 0x21U, 0x10U, 0xffU, 0xf3U, 0xd2U,
    0xcdU, 0x0cU, 0x13U, 0xecU, 0x5fU, 0x97U, 0x44U, 0x17U,
    0xc4U, 0xa7U, 0x7eU, 0x3dU, 0x64U, 0x5dU, 0x19U, 0x73U,
    0x60U, 0x81U, 0x4fU, 0xdcU, 0x22U, 0x2aU, 0x90U, 0x88U,
    0x46U, 0xeeU, 0xb8U, 0x14U, 0xdeU, 0x5eU, 0x0bU, 0xdbU,
    0xe0U, 0x32U, 0x3aU, 0x0aU, 0x49U, 0x06U, 0x24U, 0x5cU,
    0xc2U, 0xd3U, 0xacU, 0x62U, 0x91U, 0x95U, 0xe4U, 0x79U,
    0xe7U, 0xc8U, 0x37U, 0x6dU, 0x8dU, 0xd5U, 0x4eU, 0xa9U,
    0x6cU, 0x56U, 0xf4U, 0xeaU, 0x65U, 0x7aU, 0xaeU, 0x08U,
    0xbaU, 0x78U, 0x25U, 0x2eU, 0x1cU, 0xa6U, 0xb4U, 0xc6U,
    0xe8U, 0xddU, 0x74U, 0x1fU, 0x4bU, 0xbdU, 0x8bU, 0x8aU,
    0x70U, 0x3eU, 0xb5U, 0x66U, 0x48U, 0x03U, 0xf6U, 0x0eU,
    0x61U, 0x35U, 0x57U, 0xb9U, 0x86U, 0xc1U, 0x1dU, 0x9eU,
    0xe1U, 0xf8U, 0x98U, 0x11U, 0x69U, 0xd9U, 0x8eU, 0x94U,
    0x9bU, 0x1eU, 0x87U, 0xe9U, 0xceU, 0x55U, 0x28U, 0xdfU,
    0x8cU, 0xa1U, 0x89U, 0x0dU, 0xbfU, 0xe6U, 0x42U, 0x68U,
    0x41U, 0x99U, 0x2dU, 0x0fU, 0xb0U, 0x54U, 0xbbU, 0x16U
};

#define AES_XTIME(x)    ((byte)((byte)((x) << 1) ^ ((0 - ((x) >> 7)) & 0x1b)))

#define col_mul(t, i2, i3, ia, ib) \
  ( GETBYTE(t, ia) ^ GETBYTE(t, ib) ^ GETBYTE(t, i3) ^ AES_XTIME(GETBYTE(t, i2) ^ GETBYTE(t, i3)) )

#define GetTable(t, o)  t[o]
#define GetTable8(t, o) t[o]
#define GetTable_Multi(t, t0, o0, t1, o1, t2, o2, t3, o3)  \
  *(t0) = (t)[o0]; *(t1) = (t)[o1]; *(t2) = (t)[o2]; *(t3) = (t)[o3]
#define XorTable_Multi(t, t0, o0, t1, o1, t2, o2, t3, o3)  \
  *(t0) ^= (t)[o0]; *(t1) ^= (t)[o1]; *(t2) ^= (t)[o2]; *(t3) ^= (t)[o3]
#define GetTable8_4(t, o0, o1, o2, o3) \
  (((word32)(t)[o0] << 24) | ((word32)(t)[o1] << 16) |   \
   ((word32)(t)[o2] <<  8) | ((word32)(t)[o3] <<  0))

/* load sbox into cache by cache line stride */
#ifndef WOLFSSL_AES_TOUCH_LINES
    #define PreFetchSBox(x) { \
    x = 0; \
    int i; \
    for (i = 0; i < 256; i += WC_CACHE_LINE_SZ/4) { \
        x &= Tsbox[i]; \
    } \
    }
#else
    #define PreFetchSBox(x) ( x = 0 )
#endif
#endif

#if !defined(WC_AES_BITSLICED)
/* Encrypt a block using AES.
 *
 * @param [in]  aes       AES object.
 * @param [in]  inBlock   Block to encrypt.
 * @param [out] outBlock  Encrypted block.
 * @param [in]  r         Rounds divided by 2.
 * @param [in]  sz        Number of blocks to encrypt
 */
__global__ void AesEncrypt_C_CUDA(word32* rkBase, const byte* inBlockBase, byte* outBlockBase,
        word32 r, word32 sz)
{
    word32 s0, s1, s2, s3;
    word32 t0, t1, t2, t3;
    word32 sBox;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    const byte* inBlock = inBlockBase;
    byte* outBlock = outBlockBase;
    word32* rk;

    for (int i = index; i < sz; i += stride) {
        rk = rkBase;
        inBlock = inBlockBase + i * 4 * sizeof(s0);
        outBlock = outBlockBase + i * 4 * sizeof(s0);

        /*
         * map byte array block to cipher state
         * and add initial round key:
         */
        XMEMCPY(&s0, inBlock,                  sizeof(s0));
        XMEMCPY(&s1, inBlock +     sizeof(s0), sizeof(s1));
        XMEMCPY(&s2, inBlock + 2 * sizeof(s0), sizeof(s2));
        XMEMCPY(&s3, inBlock + 3 * sizeof(s0), sizeof(s3));

#ifdef LITTLE_ENDIAN_ORDER
        s0 = ByteReverseWord32(s0);
        s1 = ByteReverseWord32(s1);
        s2 = ByteReverseWord32(s2);
        s3 = ByteReverseWord32(s3);
#endif

        /* AddRoundKey */
        s0 ^= rk[0];
        s1 ^= rk[1];
        s2 ^= rk[2];
        s3 ^= rk[3];

#ifndef WOLFSSL_AES_SMALL_TABLES
#ifndef WC_NO_CACHE_RESISTANT
        s0 |= PreFetchTe();
#endif

#ifndef WOLFSSL_AES_TOUCH_LINES
#define ENC_ROUND_T_S(o)                                                       \
        t0 = GetTable(Te_CUDA[0], GETBYTE(s0, 3)) ^ GetTable(Te_CUDA[1], GETBYTE(s1, 2)) ^   \
             GetTable(Te_CUDA[2], GETBYTE(s2, 1)) ^ GetTable(Te_CUDA[3], GETBYTE(s3, 0)) ^   \
             rk[(o)+4];                                                            \
        t1 = GetTable(Te_CUDA[0], GETBYTE(s1, 3)) ^ GetTable(Te_CUDA[1], GETBYTE(s2, 2)) ^   \
             GetTable(Te_CUDA[2], GETBYTE(s3, 1)) ^ GetTable(Te_CUDA[3], GETBYTE(s0, 0)) ^   \
             rk[(o)+5];                                                            \
        t2 = GetTable(Te_CUDA[0], GETBYTE(s2, 3)) ^ GetTable(Te_CUDA[1], GETBYTE(s3, 2)) ^   \
             GetTable(Te_CUDA[2], GETBYTE(s0, 1)) ^ GetTable(Te_CUDA[3], GETBYTE(s1, 0)) ^   \
             rk[(o)+6];                                                            \
        t3 = GetTable(Te_CUDA[0], GETBYTE(s3, 3)) ^ GetTable(Te_CUDA[1], GETBYTE(s0, 2)) ^   \
             GetTable(Te_CUDA[2], GETBYTE(s1, 1)) ^ GetTable(Te_CUDA[3], GETBYTE(s2, 0)) ^   \
             rk[(o)+7]
#define ENC_ROUND_S_T(o)                                                       \
        s0 = GetTable(Te_CUDA[0], GETBYTE(t0, 3)) ^ GetTable(Te_CUDA[1], GETBYTE(t1, 2)) ^   \
             GetTable(Te_CUDA[2], GETBYTE(t2, 1)) ^ GetTable(Te_CUDA[3], GETBYTE(t3, 0)) ^   \
             rk[(o)+0];                                                            \
        s1 = GetTable(Te_CUDA[0], GETBYTE(t1, 3)) ^ GetTable(Te_CUDA[1], GETBYTE(t2, 2)) ^   \
             GetTable(Te_CUDA[2], GETBYTE(t3, 1)) ^ GetTable(Te_CUDA[3], GETBYTE(t0, 0)) ^   \
             rk[(o)+1];                                                            \
        s2 = GetTable(Te_CUDA[0], GETBYTE(t2, 3)) ^ GetTable(Te_CUDA[1], GETBYTE(t3, 2)) ^   \
             GetTable(Te_CUDA[2], GETBYTE(t0, 1)) ^ GetTable(Te_CUDA[3], GETBYTE(t1, 0)) ^   \
             rk[(o)+2];                                                            \
        s3 = GetTable(Te_CUDA[0], GETBYTE(t3, 3)) ^ GetTable(Te_CUDA[1], GETBYTE(t0, 2)) ^   \
             GetTable(Te_CUDA[2], GETBYTE(t1, 1)) ^ GetTable(Te_CUDA[3], GETBYTE(t2, 0)) ^   \
             rk[(o)+3]
#else
#define ENC_ROUND_T_S(o)                                                       \
        GetTable_Multi(Te_CUDA[0], &t0, GETBYTE(s0, 3), &t1, GETBYTE(s1, 3),            \
                              &t2, GETBYTE(s2, 3), &t3, GETBYTE(s3, 3));           \
        XorTable_Multi(Te_CUDA[1], &t0, GETBYTE(s1, 2), &t1, GETBYTE(s2, 2),            \
                              &t2, GETBYTE(s3, 2), &t3, GETBYTE(s0, 2));           \
        XorTable_Multi(Te_CUDA[2], &t0, GETBYTE(s2, 1), &t1, GETBYTE(s3, 1),            \
                              &t2, GETBYTE(s0, 1), &t3, GETBYTE(s1, 1));           \
        XorTable_Multi(Te_CUDA[3], &t0, GETBYTE(s3, 0), &t1, GETBYTE(s0, 0),            \
                              &t2, GETBYTE(s1, 0), &t3, GETBYTE(s2, 0));           \
        t0 ^= rk[(o)+4]; t1 ^= rk[(o)+5]; t2 ^= rk[(o)+6]; t3 ^= rk[(o)+7];

#define ENC_ROUND_S_T(o)                                                       \
        GetTable_Multi(Te_CUDA[0], &s0, GETBYTE(t0, 3), &s1, GETBYTE(t1, 3),            \
                              &s2, GETBYTE(t2, 3), &s3, GETBYTE(t3, 3));           \
        XorTable_Multi(Te_CUDA[1], &s0, GETBYTE(t1, 2), &s1, GETBYTE(t2, 2),            \
                              &s2, GETBYTE(t3, 2), &s3, GETBYTE(t0, 2));           \
        XorTable_Multi(Te_CUDA[2], &s0, GETBYTE(t2, 1), &s1, GETBYTE(t3, 1),            \
                              &s2, GETBYTE(t0, 1), &s3, GETBYTE(t1, 1));           \
        XorTable_Multi(Te_CUDA[3], &s0, GETBYTE(t3, 0), &s1, GETBYTE(t0, 0),            \
                              &s2, GETBYTE(t1, 0), &s3, GETBYTE(t2, 0));           \
        s0 ^= rk[(o)+0]; s1 ^= rk[(o)+1]; s2 ^= rk[(o)+2]; s3 ^= rk[(o)+3];
#endif

#ifndef WOLFSSL_AES_NO_UNROLL
    /* Unroll the loop. */
                           ENC_ROUND_T_S( 0);
        ENC_ROUND_S_T( 8); ENC_ROUND_T_S( 8);
        ENC_ROUND_S_T(16); ENC_ROUND_T_S(16);
        ENC_ROUND_S_T(24); ENC_ROUND_T_S(24);
        ENC_ROUND_S_T(32); ENC_ROUND_T_S(32);
        if (r > 5) {
            ENC_ROUND_S_T(40); ENC_ROUND_T_S(40);
            if (r > 6) {
                ENC_ROUND_S_T(48); ENC_ROUND_T_S(48);
            }
        }
        rk += r * 8;
#else
        /*
         * Nr - 1 full rounds:
         */

        for (;;) {
            ENC_ROUND_T_S(0);

            rk += 8;
            if (--r == 0) {
                break;
            }

            ENC_ROUND_S_T(0);
        }
#endif

        /*
         * apply last round and
         * map cipher state to byte array block:
         */

#ifndef WOLFSSL_AES_TOUCH_LINES
        s0 =
            (GetTable(Te_CUDA[2], GETBYTE(t0, 3)) & 0xff000000) ^
            (GetTable(Te_CUDA[3], GETBYTE(t1, 2)) & 0x00ff0000) ^
            (GetTable(Te_CUDA[0], GETBYTE(t2, 1)) & 0x0000ff00) ^
            (GetTable(Te_CUDA[1], GETBYTE(t3, 0)) & 0x000000ff) ^
            rk[0];
        s1 =
            (GetTable(Te_CUDA[2], GETBYTE(t1, 3)) & 0xff000000) ^
            (GetTable(Te_CUDA[3], GETBYTE(t2, 2)) & 0x00ff0000) ^
            (GetTable(Te_CUDA[0], GETBYTE(t3, 1)) & 0x0000ff00) ^
            (GetTable(Te_CUDA[1], GETBYTE(t0, 0)) & 0x000000ff) ^
            rk[1];
        s2 =
            (GetTable(Te_CUDA[2], GETBYTE(t2, 3)) & 0xff000000) ^
            (GetTable(Te_CUDA[3], GETBYTE(t3, 2)) & 0x00ff0000) ^
            (GetTable(Te_CUDA[0], GETBYTE(t0, 1)) & 0x0000ff00) ^
            (GetTable(Te_CUDA[1], GETBYTE(t1, 0)) & 0x000000ff) ^
            rk[2];
        s3 =
            (GetTable(Te_CUDA[2], GETBYTE(t3, 3)) & 0xff000000) ^
            (GetTable(Te_CUDA[3], GETBYTE(t0, 2)) & 0x00ff0000) ^
            (GetTable(Te_CUDA[0], GETBYTE(t1, 1)) & 0x0000ff00) ^
            (GetTable(Te_CUDA[1], GETBYTE(t2, 0)) & 0x000000ff) ^
            rk[3];
#else
    {
        word32 u0;
        word32 u1;
        word32 u2;
        word32 u3;

        s0 = rk[0]; s1 = rk[1]; s2 = rk[2]; s3 = rk[3];
        GetTable_Multi(Te_CUDA[2], &u0, GETBYTE(t0, 3), &u1, GETBYTE(t1, 3),
                              &u2, GETBYTE(t2, 3), &u3, GETBYTE(t3, 3));
        s0 ^= u0 & 0xff000000; s1 ^= u1 & 0xff000000;
        s2 ^= u2 & 0xff000000; s3 ^= u3 & 0xff000000;
        GetTable_Multi(Te_CUDA[3], &u0, GETBYTE(t1, 2), &u1, GETBYTE(t2, 2),
                              &u2, GETBYTE(t3, 2), &u3, GETBYTE(t0, 2));
        s0 ^= u0 & 0x00ff0000; s1 ^= u1 & 0x00ff0000;
        s2 ^= u2 & 0x00ff0000; s3 ^= u3 & 0x00ff0000;
        GetTable_Multi(Te_CUDA[0], &u0, GETBYTE(t2, 1), &u1, GETBYTE(t3, 1),
                              &u2, GETBYTE(t0, 1), &u3, GETBYTE(t1, 1));
        s0 ^= u0 & 0x0000ff00; s1 ^= u1 & 0x0000ff00;
        s2 ^= u2 & 0x0000ff00; s3 ^= u3 & 0x0000ff00;
        GetTable_Multi(Te_CUDA[1], &u0, GETBYTE(t3, 0), &u1, GETBYTE(t0, 0),
                              &u2, GETBYTE(t1, 0), &u3, GETBYTE(t2, 0));
        s0 ^= u0 & 0x000000ff; s1 ^= u1 & 0x000000ff;
        s2 ^= u2 & 0x000000ff; s3 ^= u3 & 0x000000ff;
    }
#endif
#else
#ifndef WC_NO_CACHE_RESISTANT
        s0 |= PreFetchSBox();
#endif

        r *= 2;
        /* Two rounds at a time */
        for (rk += 4; r > 1; r--, rk += 4) {
            t0 =
                ((word32)GetTable8(Tsbox, GETBYTE(s0, 3)) << 24) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s1, 2)) << 16) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s2, 1)) <<  8) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s3, 0)));
            t1 =
                ((word32)GetTable8(Tsbox, GETBYTE(s1, 3)) << 24) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s2, 2)) << 16) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s3, 1)) <<  8) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s0, 0)));
            t2 =
                ((word32)GetTable8(Tsbox, GETBYTE(s2, 3)) << 24) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s3, 2)) << 16) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s0, 1)) <<  8) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s1, 0)));
            t3 =
                ((word32)GetTable8(Tsbox, GETBYTE(s3, 3)) << 24) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s0, 2)) << 16) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s1, 1)) <<  8) ^
                ((word32)GetTable8(Tsbox, GETBYTE(s2, 0)));

            s0 =
                (col_mul(t0, 3, 2, 0, 1) << 24) ^
                (col_mul(t0, 2, 1, 0, 3) << 16) ^
                (col_mul(t0, 1, 0, 2, 3) <<  8) ^
                (col_mul(t0, 0, 3, 2, 1)      ) ^
                rk[0];
            s1 =
                (col_mul(t1, 3, 2, 0, 1) << 24) ^
                (col_mul(t1, 2, 1, 0, 3) << 16) ^
                (col_mul(t1, 1, 0, 2, 3) <<  8) ^
                (col_mul(t1, 0, 3, 2, 1)      ) ^
                rk[1];
            s2 =
                (col_mul(t2, 3, 2, 0, 1) << 24) ^
                (col_mul(t2, 2, 1, 0, 3) << 16) ^
                (col_mul(t2, 1, 0, 2, 3) <<  8) ^
                (col_mul(t2, 0, 3, 2, 1)      ) ^
                rk[2];
            s3 =
                (col_mul(t3, 3, 2, 0, 1) << 24) ^
                (col_mul(t3, 2, 1, 0, 3) << 16) ^
                (col_mul(t3, 1, 0, 2, 3) <<  8) ^
                (col_mul(t3, 0, 3, 2, 1)      ) ^
                rk[3];
        }

        t0 =
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 0)));
        t1 =
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 0)));
        t2 =
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 0)));
        t3 =
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 0)));
        s0 = t0 ^ rk[0];
        s1 = t1 ^ rk[1];
        s2 = t2 ^ rk[2];
        s3 = t3 ^ rk[3];
#endif

        /* write out */
#ifdef LITTLE_ENDIAN_ORDER
        s0 = ByteReverseWord32(s0);
        s1 = ByteReverseWord32(s1);
        s2 = ByteReverseWord32(s2);
        s3 = ByteReverseWord32(s3);
#endif

        XMEMCPY(outBlock,                  &s0, sizeof(s0));
        XMEMCPY(outBlock +     sizeof(s0), &s1, sizeof(s1));
        XMEMCPY(outBlock + 2 * sizeof(s0), &s2, sizeof(s2));
        XMEMCPY(outBlock + 3 * sizeof(s0), &s3, sizeof(s3));
    }
}

void AesEncrypt_C(Aes* aes, const byte* inBlock, byte* outBlock,
        word32 r)
{
    byte *inBlock_GPU = NULL;
    byte *outBlock_GPU = NULL;
    word32* rk_GPU = NULL;
    hipError_t ret = hipSuccess;

#ifdef WC_C_DYNAMIC_FALLBACK
    if ( ret == hipSuccess )
        ret = hipMalloc(&rk_GPU, sizeof(aes->key_C_fallback));
    if ( ret == hipSuccess )
        ret = hipMemcpy(rk_GPU, aes->key_C_fallback, sizeof(aes->key_C_fallback), hipMemcpyDefault);
#else
    if ( ret == hipSuccess )
        ret = hipMalloc(&rk_GPU, sizeof(aes->key));
    if ( ret == hipSuccess )
        ret = hipMemcpy(rk_GPU, aes->key, sizeof(aes->key), hipMemcpyDefault);
#endif

    if ( ret == hipSuccess )
        ret = hipMalloc(&inBlock_GPU, WC_AES_BLOCK_SIZE);
    if ( ret == hipSuccess )
        ret = hipMemcpy(inBlock_GPU, inBlock, WC_AES_BLOCK_SIZE, hipMemcpyDefault);

    if ( ret == hipSuccess )
        ret = hipMalloc(&outBlock_GPU, WC_AES_BLOCK_SIZE);

    if ( ret == hipSuccess )
        AesEncrypt_C_CUDA<<<1,1>>>(rk_GPU, inBlock_GPU, outBlock_GPU, r, 1);

    if ( ret == hipSuccess )
        ret = hipMemcpy(outBlock, outBlock_GPU, WC_AES_BLOCK_SIZE, hipMemcpyDefault);

    hipFree(inBlock_GPU);
    hipFree(outBlock_GPU);
    hipFree(rk_GPU);
}

#if defined(HAVE_AES_ECB) && !(defined(WOLFSSL_IMX6_CAAM) && \
    !defined(NO_IMX6_CAAM_AES) && !defined(WOLFSSL_QNX_CAAM))
/* Encrypt a number of blocks using AES.
 *
 * @param [in]  aes  AES object.
 * @param [in]  in   Block to encrypt.
 * @param [out] out  Encrypted block.
 * @param [in]  sz   Number of blocks to encrypt.
 */
void AesEncryptBlocks_C(Aes* aes, const byte* in, byte* out, word32 sz)
{
    byte *in_GPU = NULL;
    byte *out_GPU = NULL;
    word32* rk_GPU = NULL;
    hipError_t ret = hipSuccess;

#ifdef WC_C_DYNAMIC_FALLBACK
    if ( ret == hipSuccess )
        ret = hipMalloc(&rk_GPU, sizeof(aes->key_C_fallback));
    if ( ret == hipSuccess )
        ret = hipMemcpy(rk_GPU, aes->key_C_fallback, sizeof(aes->key_C_fallback), hipMemcpyDefault);
#else
    if ( ret == hipSuccess )
        ret = hipMalloc(&rk_GPU, sizeof(aes->key));
    if ( ret == hipSuccess )
        ret = hipMemcpy(rk_GPU, aes->key, sizeof(aes->key), hipMemcpyDefault);
#endif

    if ( ret == hipSuccess )
        ret = hipMalloc(&in_GPU, sz);
    if ( ret == hipSuccess )
        ret = hipMemcpy(in_GPU, in, sz, hipMemcpyDefault);

    if ( ret == hipSuccess )
        ret = hipMalloc(&out_GPU, sz);

    if ( ret == hipSuccess ) {
        int blockSize = 256;
        int numBlocks = (sz / WC_AES_BLOCK_SIZE + blockSize - 1) / blockSize;
        AesEncrypt_C_CUDA<<<numBlocks,blockSize>>>(rk_GPU, in_GPU, out_GPU, aes->rounds >> 1, sz / WC_AES_BLOCK_SIZE);
    }

    if ( ret == hipSuccess )
        ret = hipMemcpy(out, out_GPU, sz, hipMemcpyDefault);

    hipFree(in_GPU);
    hipFree(out_GPU);
    hipFree(rk_GPU);
}
#endif

#else

/* Encrypt a block using AES.
 *
 * @param [in]  aes       AES object.
 * @param [in]  inBlock   Block to encrypt.
 * @param [out] outBlock  Encrypted block.
 * @param [in]  r         Rounds divided by 2.
 */
__global__
void AesEncrypt_C_CUDA(Aes* aes, const byte* inBlock, byte* outBlock,
        word32 r)
{
    bs_word state[AES_BLOCK_BITS];

    (void)r;

    XMEMCPY(state, inBlock, WC_AES_BLOCK_SIZE);
    XMEMSET(((byte*)state) + WC_AES_BLOCK_SIZE, 0, sizeof(state) - WC_AES_BLOCK_SIZE);

    bs_encrypt(state, aes->bs_key, aes->rounds);

    XMEMCPY(outBlock, state, WC_AES_BLOCK_SIZE);
}

void AesEncrypt_C(Aes* aes, const byte* inBlock, byte* outBlock,
        word32 r)
{
    AesEncrypt_C_CUDA<<<1,1>>>(aes, inBlock, outBlock, r);
}

#if defined(HAVE_AES_ECB) && !(defined(WOLFSSL_IMX6_CAAM) && \
    !defined(NO_IMX6_CAAM_AES) && !defined(WOLFSSL_QNX_CAAM))
/* Encrypt a number of blocks using AES.
 *
 * @param [in]  aes  AES object.
 * @param [in]  in   Block to encrypt.
 * @param [out] out  Encrypted block.
 * @param [in]  sz   Number of blocks to encrypt.
 */
void AesEncryptBlocks_C(Aes* aes, const byte* in, byte* out, word32 sz)
{
    bs_word state[AES_BLOCK_BITS];

    while (sz >= BS_BLOCK_SIZE) {
        XMEMCPY(state, in, BS_BLOCK_SIZE);
        bs_encrypt(state, aes->bs_key, aes->rounds);
        XMEMCPY(out, state, BS_BLOCK_SIZE);
        sz  -= BS_BLOCK_SIZE;
        in  += BS_BLOCK_SIZE;
        out += BS_BLOCK_SIZE;
    }
    if (sz > 0) {
        XMEMCPY(state, in, sz);
        XMEMSET(((byte*)state) + sz, 0, sizeof(state) - sz);
        bs_encrypt(state, aes->bs_key, aes->rounds);
        XMEMCPY(out, state, sz);
    }
}
#endif

#endif /* !WC_AES_BITSLICED */

#endif /* HAVE_CUDA */

#endif /* !WOLFSSL_TI_CRYPT */

} /* extern "C" */
